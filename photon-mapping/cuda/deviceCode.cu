#include "hip/hip_runtime.h"
#include "../include/deviceCode.h"
#include "../../common/cuda/helpers.h"
#define PHOTON_ATTENUATION_FACTOR 150
#define ATTENUATE_PHOTONS false

#include <optix_device.h>

using namespace owl;

inline __device__ void savePhoton(const PhotonMapperRGD &self, PhotonMapperPRD &prd) {
  int photonIndex = atomicAdd(self.photonsCount, 1);

  auto photon = &self.photons[photonIndex];
  photon->color = prd.color;
  photon->pos = prd.scattered.origin;
  photon->dir = prd.scattered.direction;
}

inline __device__ void updateScatteredRay(Ray &ray, PhotonMapperPRD &prd) {
  ray.origin = prd.scattered.origin;
  ray.direction = prd.scattered.direction;
  prd.color = prd.scattered.color;
}

inline __device__ void shootPhoton(const PhotonMapperRGD &self, Ray &ray, PhotonMapperPRD &prd) {
  for (int i = 0; i < self.maxDepth; i++) {
    owl::traceRay(self.world, ray, prd);

    if (prd.event == SCATTER_DIFFUSE) {
      if (i > 0) savePhoton(self, prd);
      updateScatteredRay(ray, prd);
    } else {
      break;
    }
  }
}

inline __device__ void shootCausticsPhoton(const PhotonMapperRGD &self, Ray &ray, PhotonMapperPRD &prd) {
  for (int i = 0; i < self.maxDepth; i++) {
    owl::traceRay(self.world, ray, prd);

    if (i > 0 && prd.event == SCATTER_DIFFUSE) {
      savePhoton(self, prd);
    }

    if (prd.event & (SCATTER_SPECULAR | SCATTER_REFRACT)) {
      updateScatteredRay(ray, prd);
    } else {
      break;
    }
  }
}

OPTIX_RAYGEN_PROGRAM(pointLightRayGen)(){
  const auto &self = owl::getProgramData<PointLightRGD>();
  const vec2i id = owl::getLaunchIndex();

  PhotonMapperPRD prd;
  prd.random.init(id.x, id.y);
  prd.color = self.color;

  Ray ray;
  ray.origin = self.position;
  ray.direction = randomPointInUnitSphere(prd.random);
  ray.tmin = EPS;

  if (self.causticsMode) {
    shootCausticsPhoton(self, ray, prd);
  } else {
    shootPhoton(self, ray, prd);
  }
}

inline __device__ void scatterDiffuse(PhotonMapperPRD &prd, const TrianglesGeomData &self) {
  const vec3f rayDir = optixGetWorldRayDirection();
  const vec3f rayOrg = optixGetWorldRayOrigin();
  const vec3f hitPoint = rayOrg + optixGetRayTmax() * rayDir;

  const vec3f normal = getPrimitiveNormal(self);

  prd.event = SCATTER_DIFFUSE;
  prd.scattered.origin = hitPoint;
  prd.scattered.direction = reflectDiffuse(normal, prd.random);
  prd.scattered.color = multiplyColor(self.material->albedo, prd.color);
}

inline __device__ void scatterSpecular(PhotonMapperPRD &prd, const TrianglesGeomData &self) {
  const vec3f rayDir = optixGetWorldRayDirection();
  const vec3f rayOrg = optixGetWorldRayOrigin();
  const vec3f hitPoint = rayOrg + optixGetRayTmax() * rayDir;

  const vec3f normal = getPrimitiveNormal(self);

  prd.event = SCATTER_SPECULAR;
  prd.scattered.origin = hitPoint;
  prd.scattered.direction = reflect(rayDir, normal);
  prd.scattered.color = multiplyColor(self.material->albedo, prd.color);
}

inline __device__ void scatterRefract(PhotonMapperPRD &prd, const TrianglesGeomData &self) {
  const vec3f rayDir = optixGetWorldRayDirection();
  const vec3f rayOrg = optixGetWorldRayOrigin();
  const vec3f hitPoint = rayOrg + optixGetRayTmax() * rayDir;

  const vec3f normal = getPrimitiveNormal(self);

  prd.event = SCATTER_REFRACT;
  prd.scattered.origin = hitPoint;
  prd.scattered.direction = refract(rayDir, normal, self.material->refraction_idx);
  prd.scattered.color = multiplyColor(self.material->albedo, prd.color);
}

OPTIX_CLOSEST_HIT_PROGRAM(triangleMeshClosestHit)(){
  auto &prd = owl::getPRD<PhotonMapperPRD>();
  const auto &self = owl::getProgramData<TrianglesGeomData>();

  const float diffuseProb = self.material->diffuse;
  const float specularProb = self.material->specular + diffuseProb;
  const float transmissionProb = self.material->transmission + specularProb;

  const float randomProb = prd.random();
  if (randomProb < diffuseProb) {
    scatterDiffuse(prd, self);
  } else if (randomProb < specularProb) {
    scatterSpecular(prd, self);
  } else if (randomProb < transmissionProb) {
    scatterRefract(prd, self);
  } else {
    prd.event = ABSORBED;
  }
}

OPTIX_MISS_PROGRAM(miss)(){
  auto &prd = owl::getPRD<PhotonMapperPRD>();
  prd.event = MISS;
}