#include <iostream>
#include <fstream>
#include <iomanip>
// public owl node-graph API
#include "owl/owl.h"
// our device-side data structures
#include "../include/deviceCode.h"
// external helper stuff for image output
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../../common/src/assetImporter.h"
#include "../../externals/assimp/code/AssetLib/Q3BSP/Q3BSPFileData.h"
#include "../../externals/stb/stb_image_write.h"
#include "assimp/Importer.hpp"
#include "../include/program.h"
#include "../../common/src/configLoader.h"

#define LOG(message)                                            \
  std::cout << OWL_TERMINAL_BLUE;                               \
  std::cout << "#owl.sample(main): " << message << std::endl;   \
  std::cout << OWL_TERMINAL_DEFAULT;
#define LOG_OK(message)                                         \
  std::cout << OWL_TERMINAL_LIGHT_BLUE;                         \
  std::cout << "#owl.sample(main): " << message << std::endl;   \
  std::cout << OWL_TERMINAL_DEFAULT;

/* Image configuration */
auto outFileName = "result.png";

extern "C" char deviceCode_ptx[];

void writeAlivePhotons(const Photon* photons, int count, const std::string& filename) {
  std::ofstream outFile(filename);

  if (!outFile.is_open()) {
    std::cerr << "Error opening file: " << filename << std::endl;
    return;
  }

  outFile << std::fixed << std::setprecision(6);

  for (int i = 0; i < count; i++) {
    auto photon = photons[i];
    outFile << photon.pos.x << " " << photon.pos.y << " " << photon.pos.z << " "
            << photon.dir.x << " " << photon.dir.y << " " << photon.dir.z << " "
            << photon.color.x << " " << photon.color.y << " " << photon.color.z << "\n";
  }

  outFile.close();
}

void setupPointLightRayGenProgram(Program &program) {
  OWLVarDecl rayGenVars[] = {
          { "photons",OWL_BUFPTR,OWL_OFFSETOF(PointLightRGD,photons)},
          { "photonsCount",OWL_BUFPTR,OWL_OFFSETOF(PointLightRGD,photonsCount)},
          { "maxDepth",OWL_INT,OWL_OFFSETOF(PointLightRGD, maxDepth)},
          {"causticsMode", OWL_BOOL, OWL_OFFSETOF(PointLightRGD, causticsMode)},
          { "world",OWL_GROUP,OWL_OFFSETOF(PointLightRGD,world)},
          { "position",OWL_FLOAT3,OWL_OFFSETOF(PointLightRGD,position)},
          { "color",OWL_FLOAT3,OWL_OFFSETOF(PointLightRGD,color)},
          { "intensity",OWL_FLOAT,OWL_OFFSETOF(PointLightRGD,intensity)},
          { /* sentinel to mark end of list */ }
  };

  program.rayGen = owlRayGenCreate(program.owlContext,program.owlModule,"pointLightRayGen",
                                   sizeof(PointLightRGD),
                                   rayGenVars,-1);

  owlRayGenSetGroup(program.rayGen,"world",program.geometryData.worldGroup);
  owlRayGenSet1i(program.rayGen,"maxDepth",program.maxDepth);
}

void runPointLightRayGen(Program &program, const LightSource &light, bool causticsMode) {
  owlRayGenSet1b(program.rayGen,"causticsMode",causticsMode);
  owlRayGenSet3f(program.rayGen,"position",reinterpret_cast<const owl3f&>(light.pos));
  owlRayGenSet3f(program.rayGen,"color",reinterpret_cast<const owl3f&>(light.rgb));
  owlRayGenSet1f(program.rayGen,"intensity",light.power);

  if (causticsMode) {
    owlRayGenSetBuffer(program.rayGen,"photons",program.causticsPhotonsBuffer);
    owlRayGenSetBuffer(program.rayGen,"photonsCount",program.causticsPhotonsCount);
  } else {
    owlRayGenSetBuffer(program.rayGen,"photons",program.photonsBuffer);
    owlRayGenSetBuffer(program.rayGen,"photonsCount",program.photonsCount);
  }

  const int initialPhotons = light.power * (causticsMode ? program.causticsPhotonsPerWatt : program.photonsPerWatt);

  owlBuildSBT(program.owlContext);
  owlRayGenLaunch2D(program.rayGen,initialPhotons,1);
}

void initPhotonBuffers(Program &program) {
  program.photonsBuffer = owlHostPinnedBufferCreate(program.owlContext, OWL_USER_TYPE(Photon), program.castedDiffusePhotons * program.maxDepth);
  program.photonsCount = owlHostPinnedBufferCreate(program.owlContext, OWL_INT, 1);
  owlBufferClear(program.photonsCount);

  program.causticsPhotonsBuffer = owlHostPinnedBufferCreate(program.owlContext,OWL_USER_TYPE(Photon),program.castedCausticsPhotons * program.maxDepth);
  program.causticsPhotonsCount = owlHostPinnedBufferCreate(program.owlContext, OWL_INT, 1);
  owlBufferClear(program.causticsPhotonsCount);
}

void computePhotonsPerWatt(Program &program) {
  double totalWatts = 0;
  for (auto light : program.world->light_sources) {
    totalWatts += light.power;
  }

  program.photonsPerWatt = program.castedDiffusePhotons / totalWatts;
  program.causticsPhotonsPerWatt = program.castedCausticsPhotons / totalWatts;
}

void runNormal(Program &program, const std::string &output_filename) {
  LOG("launching normal photons ...")

  for (auto light : program.world->light_sources) {
    runPointLightRayGen(program, light, false);
  }

  LOG("done with launch, writing photons ...")
  auto *fb = static_cast<const Photon*>(owlBufferGetPointer(program.photonsBuffer, 0));
  auto count = *(int*)owlBufferGetPointer(program.photonsCount, 0);

  writeAlivePhotons(fb, count, output_filename);
}

void runCaustics(Program &program, const std::string &output_filename) {
  LOG("launching caustics photons ...")

  for (auto light : program.world->light_sources) {
    runPointLightRayGen(program, light, true);
  }

  LOG("done with launch, writing caustics photons ...")
  auto *fb = static_cast<const Photon*>(owlBufferGetPointer(program.causticsPhotonsBuffer, 0));
  auto count = *(int*)owlBufferGetPointer(program.causticsPhotonsCount, 0);

  writeAlivePhotons(fb, count, output_filename);
}

int main(int ac, char **av)
{
  LOG("Starting up...");

  Program program;
  program.owlContext = owlContextCreate(nullptr,1);
  program.owlModule = owlModuleCreate(program.owlContext, deviceCode_ptx);
  owlContextSetRayTypeCount(program.owlContext, 1);

  LOG("Loading Config file...")

  auto cfg = parse_config();

  auto photons_filename = cfg["data"]["photons_file"].as_string();
  auto caustics_photons_filename = cfg["data"]["caustics_photons_file"].as_string();
  auto model_path = cfg["data"]["model_path"].as_string();
  program.castedDiffusePhotons = cfg["photon-mapper"]["casted_diffuse_photons"].as_integer();
  program.castedCausticsPhotons = cfg["photon-mapper"]["casted_caustics_photons"].as_integer();
  program.maxDepth = cfg["photon-mapper"]["max_depth"].as_integer();

  auto *ai_importer = new Assimp::Importer;
  program.world =  assets::import_scene(ai_importer, model_path);

  LOG_OK("Loaded world.")

  program.geometryData = loadGeometry(program.owlContext, program.world);

  owlGeomTypeSetClosestHit(program.geometryData.trianglesGeomType, 0, program.owlModule,"triangleMeshClosestHit");
  owlMissProgCreate(program.owlContext, program.owlModule, "miss", 0, nullptr, -1);

  computePhotonsPerWatt(program);
  initPhotonBuffers(program);

  setupPointLightRayGenProgram(program);

  owlBuildPrograms(program.owlContext);
  owlBuildPipeline(program.owlContext);

  LOG("launching ...")

  runNormal(program, photons_filename);
  runCaustics(program, caustics_photons_filename);

  LOG("destroying devicegroup ...");
  owlContextDestroy(program.owlContext);

  LOG_OK("seems all went OK; app is done, this should be the last output ...");
  return 0;
}
