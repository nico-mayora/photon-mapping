#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
// public owl node-graph API
#include "owl/owl.h"
// our device-side data structures
#include "../include/deviceCode.h"
// external helper stuff for image output
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../../common/src/assetImporter.h"
#include "../../externals/stb/stb_image_write.h"
#include "../../common/src/configLoader.h"
#include <assimp/Importer.hpp>
#include "../include/program.h"
#include "../../common/src/common.h"
#include <cukd/builder.h>
#include <cukd/knn.h>
#include <chrono>

#define PHOTON_POWER (1.f)
#define CAUSTICS_PHOTON_POWER (float(PHOTON_POWER) * 0.5f)

extern "C" char deviceCode_ptx[];

Photon* readPhotonsFromFile(const std::string& filename, int& count) {
  std::ifstream file(filename);
  std::vector<Photon> tempPhotons;

  if (!file.is_open()) {
    std::cerr << "Error opening file: " << filename << std::endl;
    count = 0;
    return nullptr;
  }

  Photon photon{};
  while (file >> photon.pos.x >> photon.pos.y >> photon.pos.z
              >> photon.dir.x >> photon.dir.y >> photon.dir.z
              >> photon.color.x >> photon.color.y >> photon.color.z) {
    tempPhotons.push_back(photon);
  }

  count = tempPhotons.size();
  if (count == 0) {
    return nullptr;
  }

  auto* photonArray = new Photon[count];
  std::copy(tempPhotons.begin(), tempPhotons.end(), photonArray);

  return photonArray;
}

void loadPhotons(Program &program, const std::string& globalPhotonsFilename, const std::string& causticsPhotonsFilename) {
  int nonCausticPhotonsNum = 0;
  auto globalPhotonsFromFile = readPhotonsFromFile(globalPhotonsFilename, nonCausticPhotonsNum);
  auto causticPhotonsFromFile = readPhotonsFromFile(causticsPhotonsFilename, program.numCausticPhotons);
  program.numGlobalPhotons = nonCausticPhotonsNum + program.numCausticPhotons;
  printf("Loaded %d photons (non-caustic %d, caustic %d)\n.", program.numGlobalPhotons, nonCausticPhotonsNum, program.numCausticPhotons);

  CUKD_CUDA_CALL(MallocManaged((void **)&program.causticPhotons, program.numCausticPhotons * sizeof(Photon)));
  CUKD_CUDA_CALL(MallocManaged((void **)&program.globalPhotons,  program.numGlobalPhotons  * sizeof(Photon)));

  // Load in non Caustic photons to global map
  for (int i=0; i < nonCausticPhotonsNum; i++) {
    program.globalPhotons[i].pos = globalPhotonsFromFile[i].pos;
    program.globalPhotons[i].dir = globalPhotonsFromFile[i].dir;
    program.globalPhotons[i].color = globalPhotonsFromFile[i].color;
    program.globalPhotons[i].power = PHOTON_POWER;
  }

  // Load caustic photons to both maps
  for (int k=0; k < program.numCausticPhotons; k++) {
    program.causticPhotons[k].pos   = causticPhotonsFromFile[k].pos;
    program.causticPhotons[k].dir   = causticPhotonsFromFile[k].dir;
    program.causticPhotons[k].color = causticPhotonsFromFile[k].color;
    program.causticPhotons[k].power = CAUSTICS_PHOTON_POWER;

    program.globalPhotons[nonCausticPhotonsNum+k].pos   = causticPhotonsFromFile[k].pos;
    program.globalPhotons[nonCausticPhotonsNum+k].dir   = causticPhotonsFromFile[k].dir;
    program.globalPhotons[nonCausticPhotonsNum+k].color = causticPhotonsFromFile[k].color;
    program.globalPhotons[nonCausticPhotonsNum+k].power = CAUSTICS_PHOTON_POWER;
  }

  cukd::box_t<float3> *globalWorldBounds = NULL;
  CUKD_CUDA_CALL(MallocManaged((void **)&globalWorldBounds,sizeof(*globalWorldBounds)));

  cukd::box_t<float3> *causticWorldBounds = NULL;
  CUKD_CUDA_CALL(MallocManaged((void **)&causticWorldBounds,sizeof(*causticWorldBounds)));

  program.globalPhotonsBounds = globalWorldBounds;
  program.causticPhotonsBounds = causticWorldBounds;
  auto startKDT = std::chrono::high_resolution_clock::now();
  cukd::buildTree<Photon,Photon_traits>(program.globalPhotons,program.numGlobalPhotons, program.globalPhotonsBounds);
  cukd::buildTree<Photon,Photon_traits>(program.causticPhotons,program.numCausticPhotons, program.causticPhotonsBounds);
  auto endKDT = std::chrono::high_resolution_clock::now();
  auto durationKDT = std::chrono::duration_cast<std::chrono::milliseconds>(endKDT - startKDT);
  printf("Time taken to build KD-Tree: %d ms\n", durationKDT.count());
}
void setupCamera(Program &program, const owl::vec3f &lookFrom, const owl::vec3f &lookAt, const owl::vec3f &lookUp, float fovy) {
  const float aspect = program.frameBufferSize.x / static_cast<float>(program.frameBufferSize.y);
  const float cosFovy = std::cos(fovy);
  program.camera.pos = lookFrom;
  program.camera.dir_00 = normalize(lookAt-lookFrom);
  program.camera.dir_du = cosFovy * aspect * normalize(cross(program.camera.dir_00, lookUp));
  program.camera.dir_dv = cosFovy * normalize(cross(program.camera.dir_du, program.camera.dir_00));
  program.camera.dir_00 -= 0.5f * (program.camera.dir_du + program.camera.dir_dv);
}

void loadLights(Program &program, const std::unique_ptr<World> &world) {
  program.numLights = static_cast<int>(world->light_sources.size());
  program.lightsBuffer =  owlDeviceBufferCreate(program.owlContext, OWL_USER_TYPE(LightSource),world->light_sources.size(), world->light_sources.data());
}

void setupMissProgram(Program &program, const owl::vec3f &sky_color) {
  OWLVarDecl missProgVars[] = {
          { "sky_color", OWL_FLOAT3, OWL_OFFSETOF(MissProgData, sky_colour)},
          { /* sentinel to mark end of list */ }
  };

  auto missProg = owlMissProgCreate(program.owlContext,program.owlModule,"miss",sizeof(MissProgData),missProgVars,-1);
  owlMissProgCreate(program.owlContext,program.owlModule,"shadow",0,nullptr,-1);
  owlMissProgCreate(program.owlContext,program.owlModule,"ScatterDiffuse",0,nullptr,-1);

  owlMissProgSet3f(missProg, "sky_color", reinterpret_cast<const owl3f&>(sky_color));
}

void setupClosestHitProgram(Program &program) {
  owlGeomTypeSetClosestHit(program.geometryData.trianglesGeomType,PRIMARY,program.owlModule,"TriangleMesh");
  owlGeomTypeSetClosestHit(program.geometryData.trianglesGeomType,SHADOW,program.owlModule,"shadow");
  owlGeomTypeSetClosestHit(program.geometryData.trianglesGeomType,DIFFUSE,program.owlModule,"ScatterDiffuse");
}

void setupRaygenProgram(Program &program) {
  OWLVarDecl rayGenVars[] = {
          { "fbPtr",         OWL_BUFPTR,      OWL_OFFSETOF(RayGenData,fbPtr)},
          { "fbSize",        OWL_INT2,        OWL_OFFSETOF(RayGenData,fbSize)},
          { "world",         OWL_GROUP,       OWL_OFFSETOF(RayGenData,world)},
          { "camera.pos",    OWL_FLOAT3,      OWL_OFFSETOF(RayGenData,camera.pos)},
          { "camera.dir_00", OWL_FLOAT3,      OWL_OFFSETOF(RayGenData,camera.dir_00)},
          { "camera.dir_du", OWL_FLOAT3,      OWL_OFFSETOF(RayGenData,camera.dir_du)},
          { "camera.dir_dv", OWL_FLOAT3,      OWL_OFFSETOF(RayGenData,camera.dir_dv)},
          { "lights",        OWL_BUFPTR,      OWL_OFFSETOF(RayGenData,lights)},
          { "numLights",     OWL_INT,         OWL_OFFSETOF(RayGenData,numLights)},
          { "globalPhotons",      OWL_RAW_POINTER,  OWL_OFFSETOF(RayGenData,globalPhotons)},
          { "globalPhotonsBounds", OWL_RAW_POINTER,  OWL_OFFSETOF(RayGenData,globalPhotonsBounds)},
          { "numGlobalPhotons",   OWL_INT,          OWL_OFFSETOF(RayGenData,numGlobalPhotons)},
          { "causticPhotons",      OWL_RAW_POINTER,  OWL_OFFSETOF(RayGenData,causticPhotons)},
          { "causticPhotonsBounds", OWL_RAW_POINTER,  OWL_OFFSETOF(RayGenData,causticPhotonsBounds)},
          { "numCausticPhotons",   OWL_INT,          OWL_OFFSETOF(RayGenData,numCausticPhotons)},
          { "samples_per_pixel", OWL_INT,     OWL_OFFSETOF(RayGenData,samples_per_pixel)},
          { "max_ray_depth", OWL_INT,         OWL_OFFSETOF(RayGenData,max_ray_depth)},
          { /* sentinel to mark end of list */ }
  };

  program.rayGen = owlRayGenCreate(program.owlContext,program.owlModule,"simpleRayGen",
                           sizeof(RayGenData),
                           rayGenVars,-1);

  // ----------- set variables  ----------------------------
  owlRayGenSetBuffer(program.rayGen,"fbPtr",        program.frameBuffer);
  owlRayGenSet2i    (program.rayGen,"fbSize",       reinterpret_cast<const owl2i&>(program.frameBufferSize));
  owlRayGenSetGroup (program.rayGen,"world",        program.geometryData.worldGroup);
  owlRayGenSet3f    (program.rayGen,"camera.pos",   reinterpret_cast<const owl3f&>(program.camera.pos));
  owlRayGenSet3f    (program.rayGen,"camera.dir_00",reinterpret_cast<const owl3f&>(program.camera.dir_00));
  owlRayGenSet3f    (program.rayGen,"camera.dir_du",reinterpret_cast<const owl3f&>(program.camera.dir_du));
  owlRayGenSet3f    (program.rayGen,"camera.dir_dv",reinterpret_cast<const owl3f&>(program.camera.dir_dv));
  owlRayGenSetBuffer(program.rayGen,"lights",       program.lightsBuffer);
  owlRayGenSet1i    (program.rayGen,"numLights",    program.numLights);
  owlRayGenSetPointer(program.rayGen,"globalPhotons",     program.globalPhotons);
  owlRayGenSetPointer(program.rayGen,"globalPhotonsBounds",program.globalPhotonsBounds);
  owlRayGenSet1i    (program.rayGen,"numGlobalPhotons",   program.numGlobalPhotons);
  owlRayGenSetPointer(program.rayGen,"causticPhotons",    program.causticPhotons);
  owlRayGenSetPointer(program.rayGen,"causticPhotonsBounds",program.causticPhotonsBounds);
  owlRayGenSet1i    (program.rayGen,"numCausticPhotons",  program.numCausticPhotons);
  owlRayGenSet1i    (program.rayGen,"samples_per_pixel", program.samplesPerPixel);
  owlRayGenSet1i    (program.rayGen,"max_ray_depth", program.maxDepth);
}

int main(int ac, char **av)
{
  LOG("Starting up...")

  Program program;
  program.owlContext = owlContextCreate(nullptr,1);
  program.owlModule = owlModuleCreate(program.owlContext, deviceCode_ptx);
  owlContextSetRayTypeCount(program.owlContext, RAY_TYPES_COUNT);

  LOG("Loading Config file...")

  auto cfg = parse_config();

  auto global_photons_filename = cfg["data"]["photons_file"].as_string();
  auto caustics_photons_filename = cfg["data"]["caustics_photons_file"].as_string();
  auto model_path = cfg["data"]["model_path"].as_string();

  auto lookAt = toml_to_vec3f(cfg["camera"]["look_at"]);
  auto lookFrom = toml_to_vec3f(cfg["camera"]["look_from"]);
  auto lookUp = toml_to_vec3f(cfg["camera"]["look_up"]);
  float fovy = static_cast<float>(cfg["camera"]["fovy"].as_floating());

  auto sky_colour = toml_to_vec3f(cfg["ray-tracer"]["sky_colour"]);
  auto output_filename = cfg["ray-tracer"]["output_filename"].as_string();
  program.frameBufferSize = toml_to_vec2i(cfg["ray-tracer"]["fb_size"]);
  program.samplesPerPixel = static_cast<int>(cfg["ray-tracer"]["samples_per_pixel"].as_integer());
  program.maxDepth = static_cast<int>(cfg["ray-tracer"]["depth"].as_integer());

  auto *ai_importer = new Assimp::Importer;
  auto world =  assets::import_scene(ai_importer, model_path);

  LOG_OK("Loaded world.");

  LOG_OK("Setting up programs...");

  program.frameBuffer = owlHostPinnedBufferCreate(program.owlContext,OWL_INT,program.frameBufferSize.x * program.frameBufferSize.y);
  program.geometryData = loadGeometry(program.owlContext, world);

  loadLights(program, world);
  loadPhotons(program, global_photons_filename, caustics_photons_filename);
  setupCamera(program, lookFrom, lookAt, lookUp, fovy);

  setupMissProgram(program, sky_colour);
  setupClosestHitProgram(program);
  setupRaygenProgram(program);

  owlBuildPrograms(program.owlContext);
  owlBuildPipeline(program.owlContext);
  owlBuildSBT(program.owlContext);

  LOG_OK("Launching...");
  auto startRT = std::chrono::high_resolution_clock::now();
  owlRayGenLaunch2D(program.rayGen, program.frameBufferSize.x, program.frameBufferSize.y);
  auto endRT = std::chrono::high_resolution_clock::now();
  LOG_OK("Saving image...");

  auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endRT - startRT);
  printf("Time taken to render: %d ms\n", duration.count());

  auto *fb = static_cast<const uint32_t*>(owlBufferGetPointer(program.frameBuffer, 0));
  stbi_write_png(output_filename.c_str(),program.frameBufferSize.x,program.frameBufferSize.y,4,fb,program.frameBufferSize.x*sizeof(uint32_t));

  owlContextDestroy(program.owlContext);
  LOG_OK("Finished. If all went well, this should be the last output.");

  return 0;
}
