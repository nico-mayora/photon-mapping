#include "hip/hip_runtime.h"
#include "../include/deviceCode.h"
#include "shading.h"

#include "../../common/cuda/helpers.h"

#include <optix_device.h>
#include "owl/RayGen.h"
#include <cukd/knn.h>

#define DIRECT_LIGHT_FACTOR 1.0f
#define CAUSTICS_FACTOR 0.003f
#define DIFFUSE_FACTOR 0.005f
#define SPECULAR_FACTOR 1.f

#define NUM_DIFFUSE_SAMPLES 8

using namespace owl;

// Work-around to adding up vec3f throwing a CUDA runtime error.
struct MyColour {
  float r, g, b;
};

inline __device__
MyColour ray_colour(const RayGenData &self, Ray &ray, PerRayData &prd) {
  uint32_t p0, p1;
  packPointer(&prd, p0, p1);
  optixTrace(self.world,
    ray.origin,
    ray.direction,
    EPS,
    INFTY,
    0.f,
    OptixVisibilityMask(255),
    OPTIX_RAY_FLAG_DISABLE_ANYHIT,
    PRIMARY,
    RAY_TYPES_COUNT,
    PRIMARY,
    p0, p1
  );
  if (prd.ray_missed)
  {
    MyColour c;
    c.r = prd.colour.x;
    c.g = prd.colour.y;
    c.b = prd.colour.z;
    return c;
  }

  auto albedo = prd.hit_record.material.albedo;
  auto diffuse_brdf = prd.hit_record.material.diffuse / PI;

  // Direct light
  vec3f direct_illumination = 0.f;
  for (int l = 0; l < self.numLights; l++) {
    auto current_light = self.lights[l];

    auto shadow_ray_org = prd.hit_record.hitpoint;
    auto light_dir = current_light.pos - shadow_ray_org;
    auto distance_to_light = norm(light_dir);
    light_dir = normalize(light_dir);

    auto light_dot_norm = dot(light_dir, prd.hit_record.normal_at_hitpoint);
    if (light_dot_norm < 0.f) continue; // light hits "behind" triangle

    vec3f light_visibility = 0.f;
    uint32_t u0, u1;
    packPointer(&light_visibility, u0, u1);
    optixTrace(
      self.world,
      shadow_ray_org,
      light_dir,
      EPS,
      distance_to_light * (1.f - EPS),
      0.f,
      OptixVisibilityMask(255),
      OPTIX_RAY_FLAG_DISABLE_ANYHIT
      | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT
      | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
      SHADOW,
      RAY_TYPES_COUNT,
      SHADOW,
      u0, u1
    );

    auto specular_brdf = specularBrdf(prd.hit_record.material.specular,
      light_dir,
      ray.direction,
      prd.hit_record.normal_at_hitpoint);

    direct_illumination += light_visibility
      * static_cast<float>(current_light.power)
      * light_dot_norm
      * (1.f / (distance_to_light * distance_to_light))
      * (diffuse_brdf + specular_brdf)
      * current_light.rgb;

  }

  auto direct_term =  albedo * direct_illumination;

  MyColour final_colour;

  // Caustics
  vec3f caustics_term = gatherPhotons(prd.hit_record.hitpoint, prd.hit_record.normal_at_hitpoint, self.causticPhotons,
                                      self.numCausticPhotons, diffuse_brdf, self.causticPhotonsBounds);

  // Diffuse term
  vec3f diffuse_term = 0.f;
  #pragma unroll
  for (int s = 0; s < NUM_DIFFUSE_SAMPLES && diffuse_brdf > 0.f; s++) {
    vec3f normal = normalize(prd.hit_record.normal_at_hitpoint);

    vec3f random_vec, random_direction;
    do {
      randomUnitVector(prd.random, random_vec);
      random_direction = normal + random_vec;
    } while (nearZero(random_direction));

    random_direction = normalize(random_direction);

    PerRayData diffuse_prd;
    diffuse_prd.random.init(prd.random(), prd.random());
    diffuse_prd.ray_missed = false;
    uint32_t d0, d1;
    packPointer(&diffuse_prd, d0, d1);

    optixTrace(self.world,
      prd.hit_record.hitpoint,
      random_direction,
      EPS * 2,
      INFTY,
      0.f,
      OptixVisibilityMask(255),
      OPTIX_RAY_FLAG_DISABLE_ANYHIT,
      DIFFUSE,
      RAY_TYPES_COUNT,
      DIFFUSE,
      d0, d1
    );

    vec3f diffuse_colour = 0.f;
    if (diffuse_prd.hit_record.material.diffuse > 0.f) {
      float scattered_diffuse_brdf = diffuse_prd.hit_record.material.diffuse / PI;

      diffuse_colour = gatherPhotons(diffuse_prd.hit_record.hitpoint, diffuse_prd.hit_record.normal_at_hitpoint,
                                     self.globalPhotons, self.numGlobalPhotons, scattered_diffuse_brdf,
                                     self.globalPhotonsBounds);

      diffuse_term += diffuse_colour * diffuse_prd.hit_record.material.albedo;
    }
  }
  diffuse_term /= static_cast<float>(NUM_DIFFUSE_SAMPLES);
  diffuse_term *= albedo;

  final_colour.r = DIFFUSE_FACTOR*diffuse_term.x + CAUSTICS_FACTOR*caustics_term.x + DIRECT_LIGHT_FACTOR*direct_term.x;
  final_colour.g = DIFFUSE_FACTOR*diffuse_term.y + CAUSTICS_FACTOR*caustics_term.y + DIRECT_LIGHT_FACTOR*direct_term.y;
  final_colour.b = DIFFUSE_FACTOR*diffuse_term.z + CAUSTICS_FACTOR*caustics_term.z + DIRECT_LIGHT_FACTOR*direct_term.z;

  return final_colour;
}

inline __device__
vec3f tracePath(const RayGenData &self, Ray &ray, PerRayData &prd, const int depth) {
  vec3f colour = 0.f;
  vec3f attenuation = 1.f;
  for (int d = 0; d < depth; d++) {
    // Diffuse terms
    const auto [r, g, b] = ray_colour(self, ray, prd);
    colour += vec3f(r, g, b) * attenuation;

    bool absorbed;
    float coefficient;
    auto out_dir = reflect_or_refract_ray(
      prd.hit_record.material, ray.direction,
      prd.hit_record.normal_at_hitpoint, prd.random,
      absorbed, coefficient
    );

    if (absorbed) break;
    attenuation *= coefficient * prd.hit_record.material.albedo;

    ray = Ray(prd.hit_record.hitpoint, out_dir, EPS, INFTY);
  }

  return colour;
}

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();

  PerRayData prd;
  prd.random.init(pixelID.x,pixelID.y);

  if (pixelID.x == 600 && pixelID.y == 330)
  {
    prd.debug = true;
  }

  auto final_colour = vec3f(0.f);
  for (int sample = 0; sample < self.samples_per_pixel; sample++) {
    const auto random_eps = vec2f(prd.random(), prd.random());
    const vec2f screen = (vec2f(pixelID)+random_eps) / vec2f(self.fbSize);

    Ray ray;
    ray.origin
      = self.camera.pos;
    ray.direction
      = normalize(self.camera.dir_00
                  + screen.u * self.camera.dir_du
                  + screen.v * self.camera.dir_dv);

    const auto colour = tracePath(self, ray, prd, self.max_ray_depth);

    final_colour += colour;
  }

  final_colour = final_colour * (1.f / self.samples_per_pixel);

  const int x = pixelID.x;
  const int y = self.fbSize.y - pixelID.y;

  const int fbOfs = x+self.fbSize.x*y;

  self.fbPtr[fbOfs]
    = make_rgba(final_colour);
}

inline __device__ void closestHit() {
  auto &prd = owl::getPRD<PerRayData>();
  const auto self = owl::getProgramData<TrianglesGeomData>();

  prd.hit_record.material = *self.material;

  const vec3f rayDir = optixGetWorldRayDirection();
  const vec3f rayOrg = optixGetWorldRayOrigin();
  const auto tmax = optixGetRayTmax();

  prd.hit_record.hitpoint = rayOrg + rayDir * tmax;

  // Calculate normal at hitpoint and flip if it's pointing
  // in the same direction as the incident ray.
  const auto normal = getPrimitiveNormal(self);
  prd.hit_record.normal_at_hitpoint = (dot(rayDir, normal) < 0.f) ? normal : -normal;
  prd.hit_record.normal_at_hitpoint = normalize(prd.hit_record.normal_at_hitpoint);

  prd.colour = 0.f;
  prd.ray_missed = false;
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)() { closestHit(); }
OPTIX_CLOSEST_HIT_PROGRAM(ScatterDiffuse)() { closestHit(); }

OPTIX_MISS_PROGRAM(miss)()
{
  const MissProgData &self = owl::getProgramData<MissProgData>();

  auto &prd = owl::getPRD<PerRayData>();
  prd.colour = self.sky_colour;
  prd.ray_missed = true;
}

OPTIX_MISS_PROGRAM(ScatterDiffuse)()
{
  auto &prd = getPRD<PerRayData>();
  prd.ray_missed = true;
}

OPTIX_MISS_PROGRAM(shadow)()
{
  // we didn't hit anything, so the light is visible
  vec3f &lightVisbility = getPRD<vec3f>();
  lightVisbility = vec3f(1.f);
}

OPTIX_CLOSEST_HIT_PROGRAM(shadow)() { /* unused */}
OPTIX_ANY_HIT_PROGRAM(shadow)() { /* unused */}
