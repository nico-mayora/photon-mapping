#include "hip/hip_runtime.h"
#include "../include/deviceCode.h"
#include "../../common/cuda/helpers.h"
#include <optix_device.h>
#include "owl/RayGen.h"

using namespace owl;

#define EPS 1e-4f

OPTIX_RAYGEN_PROGRAM(photonViewerRayGen)()
{
  const auto &self = owl::getProgramData<PhotonViewerRGD>();
  const int photonId = owl::getLaunchIndex().x;
  const Photon photon = self.photons[photonId];

  if(photon.pixel.x < 0 || photon.pixel.x >= self.frameBufferSize.x ||
     photon.pixel.y < 0 || photon.pixel.y >= self.frameBufferSize.y)
  {
    //printf("x: %d, y: %d\n", photon.pixel.x, photon.pixel.y);
    return;
  }

  const auto direction = photon.pos - self.cameraPos;

  Ray ray;
  ray.origin = self.cameraPos;
  ray.direction = normalize(direction);
  ray.tmax = norm3d(direction.x, direction.y, direction.z) - EPS;

  PhotonViewerPRD prd;
  owl::traceRay(self.world, ray, prd);

  if (prd.hit){
    return;
  }

  self.frameBuffer[photon.pixel.x + self.frameBufferSize.x * photon.pixel.y] = make_rgba(photon.color);
}

OPTIX_CLOSEST_HIT_PROGRAM(photonViewerClosestHit)() {
  auto &prd = owl::getPRD<PhotonViewerPRD>();
  prd.hit = true;
}

OPTIX_MISS_PROGRAM(photonViewerMiss)()
{
  auto &prd = owl::getPRD<PhotonViewerPRD>();
  prd.hit = false;
}
